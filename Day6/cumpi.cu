#include "hip/hip_runtime.h"
//Module(s) to be loaded
//module load cuda/10.1
//module load pgi/openmpi/3.1.3/2019
//Compilation:
//nvcc cumpi.cu -I/opt/pgi/linux86-64-llvm/19.4/mpi/openmpi-3.1.3/include -L/opt/pgi/linux86-64-llvm/19.4/mpi/openmpi-3.1.3/lib -lmpi
//Execution : mpirun -n 2 ./a.out



#include<stdio.h>
#include"mpi.h"
#include"hip/hip_runtime.h"
#define N  1000

__global__ void ckrnl(int *add)
{
  int j;
  for(j=0;j<N;j++)
    add[j]=add[j]+6;
  return;
}

int main(int argc, char* argv[])
{
 int *ah,*ad;
 int i,rank,bufsize=N*4;
 MPI_Status status;

 ah = (int *)malloc(N*sizeof(int));
 hipMalloc((void **)&ad,bufsize);
 MPI_Init(&argc,&argv);
 MPI_Comm_rank(MPI_COMM_WORLD,&rank);

 if(rank == 0)
 {
  for(i=0;i<N;i++)
     ah[i]=i*5;

 hipMemcpy(ad,ah,bufsize,hipMemcpyHostToDevice);
 dim3 threads(1);
 dim3 grid(1);
 ckrnl<<< threads,grid >>>(ad);
// hipMemcpy(ah,ad,bufsize,hipMemcpyDeviceToHost);

 MPI_Send(ad,N,MPI_INT,1,100,MPI_COMM_WORLD);
}

else
{
 for(i=N-10;i<N;i++)
   printf("%2d  ",ah[i]);
 printf("\n");

 MPI_Recv(ah,N,MPI_INT,0,100,MPI_COMM_WORLD,&status);

 for(i=N-10;i<N;i++)
   printf("%2d  ",ah[i]);
 printf("\n");
}

 MPI_Finalize();
 hipFree(ad);
 return 0;
}


