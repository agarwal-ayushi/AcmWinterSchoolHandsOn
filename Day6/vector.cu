
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (65536*2048)
#define TPB 512

__global__ void mykernel(int *da, int *db, int *dc) {
	int index=threadIdx.x+blockIdx.x*blockDim.x;
	dc[index] = da[index]+db[index];
}
int main(void)
{
	int *a = new int [N]; 
	int *b = new int [N];
	int *c = new int [N];
	int *da, *db, *dc;
	int size = sizeof(int);
	
	for (int i=0; i < N; i++) {
		a[i]=i;
		b[i]=i+1;
	}
	hipMalloc((void **)&da, size*N);
	hipMalloc((void **)&db, size*N);
	hipMalloc((void **)&dc, size*N);

	hipMemcpy(da, a, size*N, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size*N, hipMemcpyHostToDevice);
	int M = N/TPB;
	mykernel<<<M, TPB>>>(da, db, dc);

	hipMemcpy(c, dc, size*N, hipMemcpyDeviceToHost);
	//printf("hellow World!");
	for (int i=0; i < 10; i++){
		//printf("%d\t", c[i]);
	}
	free(a); free(b); free(c);
	hipFree(da); hipFree(db); hipFree(dc);
	return 0;
}
