//Module(s) to be loaded
//module load cuda/10.1
//Compilation:
//nvcc mm-blas.cu -lcublas
//Execution: ./a.out

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>

#define N 15000

int main()
{
  double a[N][N] __attribute__((aligned(64)));
  double b[N][N] __attribute__((aligned(64)));
  double c[N][N] __attribute__((aligned(64)));
  double *da,*db,*dc;
  int i,j;
  printf("MM: N=%d  \n",N);

  for(i=0;i<N;i++)
   for(j=0;j<N;j++)
   {
     a[i][j]=(i%4 - j%6)*2.0;
     b[i][j]=(j%3 + i%7)*3.0;
     c[i][j]=0.0;
   }

   cublasAlloc(N*N,8,(void **) &da);
   cublasAlloc(N*N,8,(void **) &db);
   cublasAlloc(N*N,8,(void **) &dc);

   hipblasSetMatrix(N,N,8,a,N,da,N);
   hipblasSetMatrix(N,N,8,b,N,db,N);
   hipblasSetMatrix(N,N,8,c,N,dc,N);

   cublasDgemm('n','n',N,N,N,1.0,db,N,da,N,0.0,dc,N);

   hipblasGetMatrix(N,N,8,dc,N,c,N);

  printf("%lf  %lf  %lf  %lf\n",c[100][200],c[500][400],c[1000][1000],c[N-1][N-1]);
/*
  for(i=0;i<N;i++) {
   for(j=0;j<N;j++)
     printf("%lf ",a[i][j]);
     printf("\n"); }

 printf("\n");

  for(i=0;i<N;i++) {
   for(j=0;j<N;j++)
     printf("%lf ",b[i][j]);
     printf("\n"); }

 printf("\n");

  for(i=0;i<N;i++) {
   for(j=0;j<N;j++)
     printf("%lf ",c[i][j]);
     printf("\n"); }

 printf("\n");
*/
 return 0;
}

//module unload gcc/7.1.0 module load cuda/9.0  nvcc mm-blas.cu -lcublas  watch -n .1 nvidia-smi 
